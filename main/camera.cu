#include "hip/hip_runtime.h"
#include "camera.cuh"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ static void cast_ray(unsigned char *pixels, mandelbox s,
                                double VIEW_WIDTH, double VIEW_HEIGHT,
                                double WIDTH, double HEIGHT, double CANVAS_OFFSET,
                                double CAMX, double CAMY, double CAMZ,
                                double MAX_DISTANCE, int MAX_ITER, double THRESHOLD, double WORLD_SIZE) {

    // Get the position on the viewport by getting the pixel first
    unsigned int ord = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = (ord / (int) WIDTH);
    unsigned int col = (ord % (int) HEIGHT);

    double pxX = (col / WIDTH) * VIEW_WIDTH - (VIEW_WIDTH / 2);
    double pxY = (row / HEIGHT) * VIEW_HEIGHT - (VIEW_HEIGHT / 2);

    // Start at the camera position; arbitrary, but must be outside of object for best results
    double rayPX = CAMX, rayPY = CAMY, rayPZ = CAMZ;
    double rayDX = pxX, rayDY = pxY, rayDZ = CANVAS_OFFSET;
    double raySize = sqrt(rayDX*rayDX + rayDY*rayDY + rayDZ*rayDZ);

    // Normalize the direction vector
    rayDX /= raySize;
    rayDY /= raySize;
    rayDZ /= raySize;

    // March ray until threshold is reached
    double distance, totalDistance = 0;
    for (int i = 0; i < MAX_ITER; i++) {
        // Loops the world space
        // rayPX = remainder(rayPX, WORLD_SIZE);
        // rayPY = remainder(rayPY, WORLD_SIZE);
        // rayPZ = remainder(rayPZ, WORLD_SIZE);

        distance = s.distance(rayPX, rayPY, rayPZ);
        totalDistance += distance;

        if (abs(totalDistance) > MAX_DISTANCE)
            break;

        // Color the pixel when the ray is close enough and break
        if (abs(distance) < THRESHOLD) {
            double complexity = 1.0 - (i / (double) MAX_ITER);
            double distance_fade = 1.0 - pow(totalDistance / MAX_DISTANCE, 2);

            auto baseColor = (unsigned char) (255 * complexity * distance_fade);
            int p = (int) (row * 4 * WIDTH + 4 * col);
            pixels[p++] = baseColor;
            pixels[p++] = baseColor;
            pixels[p++] = baseColor;
            pixels[p++] = 255;
            return;
        }

        // Move the ray forward
        rayPX += rayDX * distance;
        rayPY += rayDY * distance;
        rayPZ += rayDZ * distance;
    }

    // Default background color
    int p = (int) (row * 4 * WIDTH + 4 * col);
    pixels[p++] = 0;
    pixels[p++] = 0;
    pixels[p++] = 0;
    pixels[p++] = 255;
}

__host__ void camera::render() const {
    // Create array of RGBA values (8-bit)
    unsigned char *pixels;

    // Compute the view plane's width and height
    const double VIEW_WIDTH = 2 * tan(FOV / 2) * CANVAS_OFFSET;
    const double VIEW_HEIGHT = VIEW_WIDTH * (HEIGHT / WIDTH);

    // Initialize a shape and create a pointer to send to kernel
    mandelbox shape = mandelbox();
    mandelbox *i = &shape;

    hipMallocManaged(&i, sizeof(shape));
    hipMallocManaged(&pixels, (int) (WIDTH * HEIGHT * 4) * sizeof(unsigned char));

    // Create threads
    cast_ray<<<262144, 64>>>(pixels, shape, VIEW_WIDTH, VIEW_HEIGHT, WIDTH, HEIGHT, CANVAS_OFFSET,
                             0, 0, -10.1000,
                             MAX_DISTANCE, MAX_ITER, THRESHOLD, WORLD_SIZE);
    hipDeviceSynchronize();

    stbi_write_png(OUT_PATH,(int) WIDTH, (int) HEIGHT,4, &pixels[0], (int) (4*WIDTH));

    hipFree(i);
    hipFree(pixels);
}


