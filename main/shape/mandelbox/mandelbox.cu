#include "hip/hip_runtime.h"
#include "mandelbox.cuh"

// DE algorithm from http://blog.hvidtfeldts.net/index.php/2011/11/distance-estimated-3d-fractals-vi-the-mandelbox/

__device__ double mandelbox::distance(double x, double y, double z) {
    double oX = x, oY = y, oZ = z;
    double *zX = &x, *zY = &y, *zZ = &z;

    double dz = 1.0;
    double *dr = &dz;

    for (int i = 0; i < 32; i++) {
        boxFold(zX, zY, zZ);
        sphereFold(zX, zY, zZ, dr);

        *zX = 2 * *zX + oX;
        *zY = 2 * *zY + oY;
        *zZ = 2 * *zZ + oZ;

        *dr = *dr * 2 + 1.0;
    }
    double r = sqrt(*zX * *zX + *zY * *zY + *zZ * *zZ);
    return r / abs(*dr);
}

__device__ void mandelbox::norm(double x, double y, double z, double *nX, double *nY, double *nZ) {

    *nX = 1;
    *nY = 0;
    *nZ = 0;
}

__device__ void mandelbox::sphereFold(double *x, double *y, double *z, double *dz) {
    double r2 = *x * *x + *y * *y + *z * *z;
    if (r2 < 0.25) {
        double temp = (.5 / 0.25);
        *x = *x * temp;
        *y = *y * temp;
        *z = *z * temp;
        *dz *= temp;
    } else if (r2 < .5) {
        double temp = (.5 / r2);
        *x = *x * temp;
        *y = *y * temp;
        *z = *z * temp;
        *dz *= temp;
    }
}

__device__ void mandelbox::boxFold(double *x, double *y, double *z) {
    *x = min(.5, max(*x, -.5)) * 2.0 - *x;
    *y = min(.5, max(*y, -.5)) * 2.0 - *y;
    *z = min(.5, max(*z, -.5)) * 2.0 - *z;
}