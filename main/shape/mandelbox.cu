#include "hip/hip_runtime.h"
#include "mandelbox.cuh"

// DE algorithm from http://blog.hvidtfeldts.net/index.php/2011/11/distance-estimated-3d-fractals-vi-the-mandelbox/

__device__ double mandelbox::distance(double x, double y, double z) {
    double oX = x, oY = y, oZ = z;
    double *zX = &x, *zY = &y, *zZ = &z;

    double dz = 1.0;
    double *dr = &dz;

    for (int i = 0; i < 8; i++) {
        boxFold(zX, zY, zZ);
        sphereFold(zX, zY, zZ, dr);

        *zX = renderScale * *zX + oX;
        *zY = renderScale * *zY + oY;
        *zZ = renderScale * *zZ + oZ;

        *dr = *dr * abs(renderScale) + 1.0;
    }
    return sqrt(*zX * *zX + *zY * *zY + *zZ * *zZ) / abs(*dr);
}

__device__ void mandelbox::sphereFold(double *x, double *y, double *z, double *dz) {
    double r2 = *x * *x + *y * *y + *z * *z;
    if (r2 < 0.25) {
        double temp = (.5 / 0.25);
        *x = *x * temp;
        *y = *y * temp;
        *z = *z * temp;
        *dz *= temp;
    } else if (r2 < .5) {
        double temp = (.5 / r2);
        *x = *x * temp;
        *y = *y * temp;
        *z = *z * temp;
        *dz *= temp;
    }
}

__device__ void mandelbox::boxFold(double *x, double *y, double *z) {
    *x = min(.5, max(*x, -.5)) * 2.0 - *x;
    *y = min(.5, max(*y, -.5)) * 2.0 - *y;
    *z = min(.5, max(*z, -.5)) * 2.0 - *z;
}